#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void addIntsCUDA(int *a, int *b) {
    a[0] += b[0];
}

int main() {
    int a = 5;
    int b = 9;
    int res;
    int *d_a, *d_b;

    // Allocate memory for the kernel function in the GPU
    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));

    // Copy the memory from the CPU to the GPU
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    // Execute the kernel function
    addIntsCUDA<<<1, 1>>>(d_a, d_b);

    // Copy the result from the GPU to the CPU
    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    cout<<"The result is "<< a <<endl;

    // Clear the GPU memory
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
