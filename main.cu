#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
 
// CUDA kernel to add elements of two arrays
__global__ void AplusB(int *res, int a, int b) {
    res[threadIdx.x] = a + b + threadIdx.x;
    printf("GPU: %d: A+B = %d\n", threadIdx.x, res[threadIdx.x]);
}
 
int main(void)
{
    int N = 1000;
    int *res;
    int *x, *y;

    // Allocate memory
    hipMallocManaged(&res, 1000 * sizeof(int));

    // Execute kernel function
    AplusB<<< 1, N >>>(res, 10, 100);
    hipError_t err = hipPeekAtLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
    }

    // Wait until GPU completes calculations
    hipDeviceSynchronize();

    // // Print result
    // for(int i = 0; i < N; i++) {
    //     printf("CPU: %d: A+B = %d\n", i, res[i]);
    // }

    // Clear memory
    hipFree(res);

    return 0;
}