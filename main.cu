#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void addIntsCUDA(int *a, int *b) {
    a[0] += b[0];
}

int main() {
    int a = 5;
    int b = 9;
    // int res;
    int *d_a, *d_b;

    d_a[0] = a;
    d_b[0] = b;

    // Allocate memory for the kernel function in the GPU
    hipMallocManaged(&d_a, sizeof(int));
    hipMallocManaged(&d_b, sizeof(int));

    // // Allocate memory for the kernel function in the GPU
    // cudaMalloc(&d_a, sizeof(int));
    // cudaMalloc(&d_b, sizeof(int));

    // // Copy the memory from the CPU to the GPU
    // cudaMemcpy(d_a, &a, sizeof(int), cudaMemcpyHostToDevice);
    // cudaMemcpy(d_b, &b, sizeof(int), cudaMemcpyHostToDevice);

    // Execute the kernel function
    addIntsCUDA<<<1, 1>>>(d_a, d_b);

    // // Copy the result from the GPU to the CPU
    // cudaMemcpy(&a, d_a, sizeof(int), cudaMemcpyDeviceToHost);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Print the result
    cout<<"The result is "<< a <<endl;

    // Clear the GPU memory
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
