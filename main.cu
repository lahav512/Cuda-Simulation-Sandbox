#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
 
// CUDA kernel to add elements of two arrays
__global__ void AplusB(int *res, int a, int b) {
    res[threadIdx.x] = a + b + threadIdx.x;
}
 
int main(void)
{
    int N = 1000;
    int *res;

    // Allocate memory
    hipMallocManaged(&res, 1000 * sizeof(int));

    // Execute kernel function
    AplusB<<< 1, N >>>(res, 10, 100);

    // Wait until GPU completes calculations
    hipDeviceSynchronize();

    // Print result
    for(int i = 0; i < N; i++) {
        printf("%d: A+B = %d\n", i, res[i]);
    }

    // Clear memory
    hipFree(res);

    return 0;
}