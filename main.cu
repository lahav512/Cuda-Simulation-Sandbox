#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
 
// CUDA kernel to add elements of two arrays
__global__ void add(int n, float *x, float *y) {
    for (int i = index; i < n; i += 1) {
        y[i] += x[i];
    }
}
 
int main(void)
{
    int N = 1;
    float *x, *y;

    // Allocate Unified Memory -- accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Launch kernel on 1M elements on the GPU
    add<<<1, 1>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Print result
    for (int i = 0; i < N; i++) {
        std::cout << "x + y = " << y[i] << std::endl;
    }

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}